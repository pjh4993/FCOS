#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCDeviceUtils.cuh>

#include <vector>
#include <iostream>
#define BOX_DIM 8
/*
a[0 ~ 3]  : regressed box left, top, right, bottom
a[4]      : confidence of box
a[5]      : label
a[6 ~ 7]  : loction of regressed posiiton (x, y)
a[8]      : centerness of box
*/

int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float devIoU(float const * const a, float const * const b) {
  if (a[5] != b[5]) {
    return 0.0;
  }
  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

__device__ inline int devPredLoc(float const * const a, float const * const b) {
  if (a[5] != b[5]){
    return 0;
  }
  /*
  //a box's center and b box's center
  //check centerness of each other and if they are same they are detecting same object
  float a_hor = (a[0] + a[2])/2, a_ver = (a[1] + a[3])/2;
  float b_hor = (b[0] + b[2])/2, b_ver = (b[1] + b[3])/2;

  float center_dist = (a_hor - b_hor)*(a_hor - b_hor) + (a_ver - b_ver) * (a_ver - b_ver);
  center_dist = sqrt(center_dist);
  
  return 0;
  */

  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  int retA=1, retB = 1;
  if(a[6] > left && a[6] < right && a[7] < bottom && a[7] > top){
    retA = 0;
  }
  if(b[6] > left && b[6] < right && b[7] < bottom && b[7] > top){
    retB = 0;
  }
  //return 0 -> nms out, return 1 -> nms stay ???
  return 1;
  return (retA + retB) == 0;
}

__global__ void ml_nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * BOX_DIM];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * BOX_DIM + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * BOX_DIM + 0];
    block_boxes[threadIdx.x * BOX_DIM + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * BOX_DIM + 1];
    block_boxes[threadIdx.x * BOX_DIM + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * BOX_DIM + 2];
    block_boxes[threadIdx.x * BOX_DIM + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * BOX_DIM + 3];
    block_boxes[threadIdx.x * BOX_DIM + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * BOX_DIM + 4];
    block_boxes[threadIdx.x * BOX_DIM + 5] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * BOX_DIM + 5];
    block_boxes[threadIdx.x * BOX_DIM + 6] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * BOX_DIM + 6];
    block_boxes[threadIdx.x * BOX_DIM + 7] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * BOX_DIM + 7];


  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * BOX_DIM;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if ((devIoU(cur_box, block_boxes + i * BOX_DIM) > nms_overlap_thresh) && devPredLoc(cur_box, block_boxes + i* BOX_DIM)) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = THCCeilDiv(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

// boxes is a N x BOX_DIM tensor
at::Tensor ml_nms_cuda(const at::Tensor boxes, float nms_overlap_thresh) {
  using scalar_t = float;
  AT_ASSERTM(boxes.type().is_cuda(), "boxes must be a CUDA tensor");
  auto scores = boxes.select(1, 4);
  auto order_t = std::get<1>(scores.sort(0, /* descending=*/true));
  auto boxes_sorted = boxes.index_select(0, order_t);

  int boxes_num = boxes.size(0);

  const int col_blocks = THCCeilDiv(boxes_num, threadsPerBlock);

  scalar_t* boxes_dev = boxes_sorted.data<scalar_t>();

  THCState *state = at::globalContext().lazyInitCUDA(); // TODO replace with getTHCState

  unsigned long long* mask_dev = NULL;
  //THCudaCheck(THCudaMalloc(state, (void**) &mask_dev,
  //                      boxes_num * col_blocks * sizeof(unsigned long long)));

  mask_dev = (unsigned long long*) THCudaMalloc(state, boxes_num * col_blocks * sizeof(unsigned long long));

  dim3 blocks(THCCeilDiv(boxes_num, threadsPerBlock),
              THCCeilDiv(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
  ml_nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);

  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  THCudaCheck(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  at::Tensor keep = at::empty({boxes_num}, boxes.options().dtype(at::kLong).device(at::kCPU));
  int64_t* keep_out = keep.data<int64_t>();

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long *p = &mask_host[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }

  THCudaFree(state, mask_dev);
  // TODO improve this part
  return std::get<0>(order_t.index({
                       keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep).to(
                         order_t.device(), keep.scalar_type())
                     }).sort(0, false));
}
